#include "hip/hip_runtime.h"
#include <kernel.hpp>
#include <Eigen/Core>

#include <iostream>
#include <stdio.h>

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

// CUDA Version
namespace Kernel
{
    // __device__ void getSeparatingHyperplane(double *obstacle_A, double *obstacle_b, double *obstacle_Adj, double *obstacle_v, const double *x, double *A_hyp, double &b_hyp, double &dist)
    // {
    //     int closest_point = -1;
    //     double closest_dist = 1e3;
    //     double dist_to_point = 0.0;
    //     double temp[2] = {0.0};

    //     for (int j = 0; j < 4; j++) {
    //         dist_to_point = 0.0;
    //         for (int k = 0; k < 2; k++) {
    //             temp[k] = x[k] - obstacle_v[j + k*4];
    //             dist_to_point += temp[k] * temp[k];
    //         }
    //         if (dist_to_point < closest_dist) {
    //             closest_point = j;
    //             closest_dist = dist_to_point;
    //         }
    //     }
    //     dist = closest_dist;

    //     const int num_faces = 4;
    //     for (int j = 0; j < num_faces; j++) {
    //         if ((obstacle_Adj[closest_point + j * 4] > 0) && (obstacle_A[j] * x[0] + obstacle_A[j + 4] * x[1] - obstacle_b[j] > -1e-2)) {
    //             for (int k = 0; k < 2; k++) {
    //                 A_hyp[k] += obstacle_A[j + 4*k];
    //             }
    //         }
    //     }

    //     double norm = sqrt(A_hyp[0] * A_hyp[0] + A_hyp[1] * A_hyp[1]);
    //     A_hyp[0] /= norm;
    //     A_hyp[1] /= norm;
    //     b_hyp = A_hyp[0] * obstacle_v[closest_point] + A_hyp[1] * obstacle_v[closest_point + 4];
    // }

        __device__ void getSeparatingHyperplane(double *obstacle_A, double *obstacle_b, double *obstacle_Adj, double *obstacle_v, const double *x, double *A_hyp, double &b_hyp, double &dist)
    {
        int closest_point = -1;
        double closest_dist = 1e3;
        double dist_to_point = 0;
        for (int j = 0; j < 4; j++) {
            dist_to_point = (x[0] - obstacle_v[j]) * (x[0] - obstacle_v[j]) +  (x[1] - obstacle_v[j + 4])*(x[1] - obstacle_v[j + 4]);
            if (dist_to_point < closest_dist) {
                closest_point = j;
                closest_dist = dist_to_point;
            }
        }
        dist = closest_dist;

        int num_constraint_violated = 0;
        const int num_faces = 4;
        for (int j = 0; j < num_faces; j++) {
            if ((obstacle_Adj[closest_point + j * 4] > 0) && (obstacle_A[j] * x[0] + obstacle_A[j + 4] * x[1] - obstacle_b[j] > -1e-2)) {
                A_hyp[0] = obstacle_A[j + 4*0];
                A_hyp[1] = obstacle_A[j + 4*1];
                num_constraint_violated++;
            }
        }
        if (num_constraint_violated == 0) {
            // printf("Constraint Violated . . . . . ");
            A_hyp[0] = 0;
            A_hyp[1] = 0;
            b_hyp = 1;
        } else {
            if (num_constraint_violated > 1) {
                A_hyp[0] = x[0] - obstacle_v[closest_point];
                A_hyp[1] = x[1] - obstacle_v[closest_point + 4];
            }
            double norm = sqrt(A_hyp[0] * A_hyp[0] + A_hyp[1] * A_hyp[1]);
            A_hyp[0] /= norm;
            A_hyp[1] /= norm;
            b_hyp = A_hyp[0] * obstacle_v[closest_point] + A_hyp[1] * obstacle_v[closest_point + 4];   
        }
    }

    __global__ void obstacleMembershipHeuristic(double *obstacle_A, double *obstacle_b, double *obstacle_Adj, double *obstacle_v, const double *edges, int *member, const int num_edges, const int num_obstacles)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= num_edges * num_obstacles)
            return;

        const int edge_number = i % num_edges;
        const int obstacle_number = i / num_edges;

        const double *edge = &edges[edge_number * 16];
        double *A = &obstacle_A[obstacle_number *  16];
        double *b = &obstacle_b[obstacle_number * 4];
        double *Adj = &obstacle_Adj[obstacle_number * 16];
        double *v = &obstacle_v[obstacle_number * 8];

        bool in_obstacle = false;
        for (int k = 0; k < 4; k++) {
            bool result[4] = {false};
            for (int j = 0; j < 4; j++)
            {
                result[j] = (A[j] * edge[4*k] + A[j + 4] * edge[4*k+1] + A[j + 8] * edge[4*k+2] + A[j + 12] * edge[4*k+3] - b[j]) <= 0;
            }
            if (result[0] & result[1] & result[2] & result[3])
            {
                in_obstacle = true;
                break;
            }
        }
        if (in_obstacle)
        {
            member[i] = 1;
        }
        else
        {
            // If not in obstacle, then we have to see if we have a separating hyperplane
            double A_hyp[2] = {0.0};
            double b_hyp = 0.0;

            for (int j = 0; j < 4; j++) {
                double A_hyp_[2] {0.0};
                double b_hyp_ = 0.0;
                double dist;
                getSeparatingHyperplane(A, b, Adj, v, &edge[j * 4], A_hyp_, b_hyp_, dist);
                A_hyp[0] += A_hyp_[0];
                A_hyp[1] += A_hyp_[1];
                b_hyp += b_hyp_;
            }

            A_hyp[0] /= 4;
            A_hyp[1] /= 4;
            b_hyp /= 4;

            bool safe = true;
            for (int j = 0; j < 4; j++) {
                double result = A_hyp[0] * edge[j * 4] + A_hyp[1] * edge[j * 4 + 1] - b_hyp;
                if (result < 0) {
                    safe = false;
                    break;
                }
            }

            member[i] = safe ? 0 : 2;

        }
    }

    void GraphQP_ObstacleMembershipHeuristic(std::vector<Obstacle> obstacles, const std::vector<matrix_t> &edges, int_vector_t &member)
    {
        int num_edges = edges.size();
        int num_obstacles = obstacles.size();
        int member_size = num_obstacles * num_edges * sizeof(int);

        // Memory sizes
        size_t obstacle_A_size = num_obstacles * 16 * sizeof(double);        // 4x4 matrix
        size_t obstacle_b_size = num_obstacles * 4 * sizeof(double);         // 4x1 vector
        size_t obstacle_Adj_size = num_obstacles * 16 * sizeof(double);         // 4x1 vector
        size_t obstacle_v_size = num_obstacles * 8 * sizeof(double);         // 4x1 vector
        size_t edges_size = num_edges * 16 * sizeof(double); // num_edges x 4x4 matrix

        // Allocate memory on the device
        double *d_obstacle_A;
        double *d_obstacle_b;
        double *d_obstacle_Adj;
        double *d_obstacle_v;
        double *d_edges;
        int *d_member;

        hipMalloc((void **)&d_obstacle_A, obstacle_A_size);
        hipMalloc((void **)&d_obstacle_b, obstacle_b_size);
        hipMalloc((void **)&d_obstacle_Adj, obstacle_Adj_size);
        hipMalloc((void **)&d_obstacle_v, obstacle_v_size);
        hipMalloc((void **)&d_edges, edges_size);
        hipMalloc((void **)&d_member, member_size);

        // Prepare edge data
        double obstacle_A_flat[num_obstacles * 16];
        double obstacle_b_flat[num_obstacles * 4];
        double obstacle_Adj_flat[num_obstacles * 16];
        double obstacle_v_flat[num_obstacles * 8];
        double edges_flat[num_edges * 16];

        for (int o = 0; o < num_obstacles; o++) {
            // copy obstacle A
            for (int col = 0; col < 4; col++)
            {
                for (int row = 0; row < 4; row++)
                {
                    obstacle_A_flat[o * 16 + row + col*4] = obstacles[o].A(row, col);
                    obstacle_Adj_flat[o * 16 + row + col*4] = obstacles[o].Adjacency(row, col);
                }
                obstacle_b_flat[o * 4 + col] = obstacles[o].b(col);
            }
            for (int col = 0; col < 2; col++)
            {
                for (int row = 0; row < 4; row++)
                {
                    obstacle_v_flat[o * 8 + row + col*4] = obstacles[o].v(row, col);
                }
            }
        }
        // copy edges
        for (int i = 0; i < num_edges; i++)
        {
            Eigen::MatrixXd mat = edges[i];
            for (int col = 0; col < 4; col++)
            {
                for (int row = 0; row < 4; row++)
                {
                    edges_flat[i * 16 + row + col*4] = mat(row, col);
                }
            }
        }

        // hipEvent_t start, stop;
        // hipEventCreate(&start);
        // hipEventCreate(&stop);

        // Copy data to device
        hipMemcpy(d_obstacle_A, obstacle_A_flat, obstacle_A_size, hipMemcpyHostToDevice);
        hipMemcpy(d_obstacle_b, obstacle_b_flat, obstacle_b_size, hipMemcpyHostToDevice);
        hipMemcpy(d_obstacle_Adj, obstacle_Adj_flat, obstacle_Adj_size, hipMemcpyHostToDevice);
        hipMemcpy(d_obstacle_v, obstacle_v_flat, obstacle_v_size, hipMemcpyHostToDevice);
        hipMemcpy(d_edges, edges_flat, edges_size, hipMemcpyHostToDevice);

        // Launch the kernel
        // CAUTION: THIS CANNOT BE MORE THAN YOUR TENSOR CORE COUNT
        int blockSize = 128;
        int gridSize = (num_obstacles * num_edges + blockSize - 1) / blockSize;

        // hipEventRecord(start);
        obstacleMembershipHeuristic<<<gridSize, blockSize>>>(d_obstacle_A, d_obstacle_b, d_obstacle_Adj, d_obstacle_v, d_edges, d_member, num_edges, num_obstacles);
        // hipEventRecord(stop);

        // Copy the result back to the host
        hipMemcpy(member.data(), d_member, member_size, hipMemcpyDeviceToHost);

        // hipEventSynchronize(stop);
        // float milliseconds = 0;
        // hipEventElapsedTime(&milliseconds, start, stop);
        // printf("That took: %f ms\n", milliseconds);

        // Free device memory
        hipFree(d_obstacle_A);
        hipFree(d_obstacle_b);
        hipFree(d_obstacle_Adj);
        hipFree(d_obstacle_v);
        hipFree(d_edges);
        hipFree(d_member);
    }

    __global__ void getAllHyperplanes(double *obstacle_A, double *obstacle_b, double *obstacle_Adj, double *obstacle_v, const double *sol, double *A1, double *A2, double *b, double *dist, const int N, const int num_obstacles)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= N * num_obstacles)
            return;

        const int sol_number = i % N;
        const int obstacle_number = i / N;

        const double *sol_k = &sol[sol_number * 4];
        double *A_obst = &obstacle_A[obstacle_number *  16];
        double *b_obst = &obstacle_b[obstacle_number * 4];
        double *Adj_obst = &obstacle_Adj[obstacle_number * 16];
        double *v_obst = &obstacle_v[obstacle_number * 8];
        
        double A_hyp[2] = {0.0};
        double b_hyp = 0.0;
        double the_dist = 1e3;

        getSeparatingHyperplane(A_obst, b_obst, Adj_obst, v_obst, sol_k, A_hyp, b_hyp, the_dist);
        A1[i] = A_hyp[0];
        A2[i] = A_hyp[1];
        b[i] = b_hyp;
        dist[i] = the_dist;

    }

    void MPC_GetActiveConstraints(std::vector<Obstacle> obstacles, const vector_t &sol, vector_t &A1, vector_t &A2, vector_t &b, vector_t &dist)
    {
        int num_obstacles = obstacles.size();
        int N = sol.size() / 4;

        // Memory sizes
        size_t obstacle_A_size = num_obstacles * 16 * sizeof(double);        // 4x4 matrix
        size_t obstacle_b_size = num_obstacles * 4 * sizeof(double);         // 4x1 vector
        size_t obstacle_Adj_size = num_obstacles * 16 * sizeof(double);         // 4x4 matrix
        size_t obstacle_v_size = num_obstacles * 8 * sizeof(double);         // 4x2 matrix
        size_t sol_size = sol.size() * sizeof(double);
        size_t A1_size = A1.size() * sizeof(double); 
        size_t A2_size = A2.size() * sizeof(double); 
        size_t b_size = b.size() * sizeof(double); 
        size_t dist_size = dist.size() * sizeof(double); 

        // Allocate memory on the device
        double *d_obstacle_A;
        double *d_obstacle_b;
        double *d_obstacle_Adj;
        double *d_obstacle_v;
        double *d_sol;
        double *d_A1;
        double *d_A2;
        double *d_b;
        double *d_dist;

        hipMalloc((void **)&d_obstacle_A, obstacle_A_size);
        hipMalloc((void **)&d_obstacle_b, obstacle_b_size);
        hipMalloc((void **)&d_obstacle_Adj, obstacle_Adj_size);
        hipMalloc((void **)&d_obstacle_v, obstacle_v_size);
        hipMalloc((void **)&d_sol, sol_size);
        hipMalloc((void **)&d_A1, A1_size);
        hipMalloc((void **)&d_A2, A2_size);
        hipMalloc((void **)&d_b, b_size);
        hipMalloc((void **)&d_dist, dist_size);

        // Prepare edge data
        double obstacle_A_flat[num_obstacles * 16];
        double obstacle_b_flat[num_obstacles * 4];
        double obstacle_Adj_flat[num_obstacles * 16];
        double obstacle_v_flat[num_obstacles * 8];

        for (int o = 0; o < num_obstacles; o++) {
            // copy obstacle A
            for (int col = 0; col < 4; col++)
            {
                for (int row = 0; row < 4; row++)
                {
                    obstacle_A_flat[o * 16 + row + col*4] = obstacles[o].A(row, col);
                    obstacle_Adj_flat[o * 16 + row + col*4] = obstacles[o].Adjacency(row, col);
                }
                obstacle_b_flat[o * 4 + col] = obstacles[o].b(col);
            }
            for (int col = 0; col < 2; col++)
            {
                for (int row = 0; row < 4; row++)
                {
                    obstacle_v_flat[o * 8 + row + col*4] = obstacles[o].v(row, col);
                }
            }
        }

        // hipEvent_t start, stop;
        // hipEventCreate(&start);
        // hipEventCreate(&stop);

        // Copy data to device
        hipMemcpy(d_obstacle_A, obstacle_A_flat, obstacle_A_size, hipMemcpyHostToDevice);
        hipMemcpy(d_obstacle_b, obstacle_b_flat, obstacle_b_size, hipMemcpyHostToDevice);
        hipMemcpy(d_obstacle_Adj, obstacle_Adj_flat, obstacle_Adj_size, hipMemcpyHostToDevice);
        hipMemcpy(d_obstacle_v, obstacle_v_flat, obstacle_v_size, hipMemcpyHostToDevice);
        hipMemcpy(d_sol, sol.data(), sol_size, hipMemcpyHostToDevice);

        // Launch the kernel
        // CAUTION: THIS CANNOT BE MORE THAN YOUR TENSOR CORE COUNT
        int blockSize = 128;
        int gridSize = (num_obstacles * N + blockSize - 1) / blockSize;

        // hipEventRecord(start);
        getAllHyperplanes<<<gridSize, blockSize>>>(d_obstacle_A, d_obstacle_b, d_obstacle_Adj, d_obstacle_v, d_sol, d_A1, d_A2, d_b, d_dist, N, num_obstacles);
        // hipEventRecord(stop);

        // Copy the result back to the host
        hipMemcpy(A1.data(), d_A1, A1_size, hipMemcpyDeviceToHost);
        hipMemcpy(A2.data(), d_A2, A2_size, hipMemcpyDeviceToHost);
        hipMemcpy(b.data(), d_b, b_size, hipMemcpyDeviceToHost);
        hipMemcpy(dist.data(), d_dist, dist_size, hipMemcpyDeviceToHost);

        // hipEventSynchronize(stop);
        // float milliseconds = 0;
        // hipEventElapsedTime(&milliseconds, start, stop);
        // printf("That took: %f ms\n", milliseconds);

        // Free device memory
        hipFree(d_obstacle_A);
        hipFree(d_obstacle_b);
        hipFree(d_obstacle_Adj);
        hipFree(d_obstacle_v);
        hipFree(d_sol);
        hipFree(d_A1);
        hipFree(d_A2);
        hipFree(d_b);
        hipFree(d_dist);
    }

}
