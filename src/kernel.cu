#include "hip/hip_runtime.h"
#include <kernel.hpp>
#include <Eigen/Core>

#include <iostream>
#include <stdio.h>

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

// CUDA Version
namespace Kernel
{
    __device__ void getSeparatingHyperplane(double *obstacle_A, double *obstacle_b, double *obstacle_Adj, double *obstacle_v, const double *x, double *A_hyp, double &b_hyp)
    {
        int closest_point = -1;
        double closest_dist = 1e3;
        double dist_to_point = 0.0;
        double temp[2] = {0.0};

        for (int j = 0; j < 4; j++) {
            dist_to_point = 0.0;
            for (int k = 0; k < 2; k++) {
                temp[k] = x[k] - obstacle_v[j + k*4];
                dist_to_point += temp[k] * temp[k];
            }
            if (dist_to_point < closest_dist) {
                closest_point = j;
                closest_dist = dist_to_point;
            }
        }

        const int num_faces = 4;
        for (int j = 0; j < num_faces; j++) {
            if ((obstacle_Adj[closest_point + j * 4] > 0) && (obstacle_A[j] * x[0] + obstacle_A[j + 4] * x[1] - obstacle_b[j] > -1e-2)) {
                for (int k = 0; k < 2; k++) {
                    A_hyp[k] += obstacle_A[j + 4*k];
                }
            }
        }

        double norm = sqrt(A_hyp[0] * A_hyp[0] + A_hyp[1] * A_hyp[1]);
        A_hyp[0] /= norm;
        A_hyp[1] /= norm;
        b_hyp = A_hyp[0] * obstacle_v[closest_point] + A_hyp[1] * obstacle_v[closest_point + 4];
    }

    __global__ void obstacleMembershipHeuristic(double *obstacle_A, double *obstacle_b, double *obstacle_Adj, double *obstacle_v, const double *edges, int *member, const int num_edges, const int num_obstacles)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= num_edges * num_obstacles)
            return;

        const int edge_number = i % num_edges;
        const int obstacle_number = i / num_edges;

        const double *edge = &edges[edge_number * 16];
        double *A = &obstacle_A[obstacle_number *  16];
        double *b = &obstacle_b[obstacle_number * 4];
        double *Adj = &obstacle_Adj[obstacle_number * 16];
        double *v = &obstacle_v[obstacle_number * 8];

        bool in_obstacle = false;
        for (int k = 0; k < 4; k++) {
            bool result[4] = {false};
            for (int j = 0; j < 4; j++)
            {
                result[j] = (A[j] * edge[4*k] + A[j + 4] * edge[4*k+1] + A[j + 8] * edge[4*k+2] + A[j + 12] * edge[4*k+3] - b[j]) <= 0;
            }
            if (result[0] & result[1] & result[2] & result[3])
            {
                in_obstacle = true;
                break;
            }
        }
        if (in_obstacle)
        {
            member[i] = 1;
        }
        else
        {
            // If not in obstacle, then we have to see if we have a separating hyperplane
            double A_hyp[2] = {0.0};
            double b_hyp = 0.0;

            for (int j = 0; j < 4; j++) {
                double A_hyp_[2] {0.0};
                double b_hyp_ = 0.0;
                getSeparatingHyperplane(A, b, Adj, v, &edge[j * 4], A_hyp_, b_hyp_);
                A_hyp[0] += A_hyp_[0];
                A_hyp[1] += A_hyp_[1];
                b_hyp += b_hyp_;
            }

            A_hyp[0] /= 4;
            A_hyp[1] /= 4;
            b_hyp /= 4;

            bool safe = true;
            for (int j = 0; j < 4; j++) {
                double result = A_hyp[0] * edge[j * 4] + A_hyp[1] * edge[j * 4 + 1] - b_hyp;
                if (result < 0) {
                    safe = false;
                    break;
                }
            }

            member[i] = safe ? 0 : 2;

        }
    }

    void GraphQP_ObstacleMembershipHeuristic(std::vector<Obstacle> obstacles, const std::vector<matrix_t> &edges, int_vector_t &member)
    {
        int num_edges = edges.size();
        int num_obstacles = obstacles.size();
        int member_size = num_obstacles * num_edges * sizeof(int);

        // Memory sizes
        size_t obstacle_A_size = num_obstacles * 16 * sizeof(double);        // 4x4 matrix
        size_t obstacle_b_size = num_obstacles * 4 * sizeof(double);         // 4x1 vector
        size_t obstacle_Adj_size = num_obstacles * 16 * sizeof(double);         // 4x1 vector
        size_t obstacle_v_size = num_obstacles * 8 * sizeof(double);         // 4x1 vector
        size_t edges_size = num_edges * 16 * sizeof(double); // num_edges x 4x4 matrix

        // Allocate memory on the device
        double *d_obstacle_A;
        double *d_obstacle_b;
        double *d_obstacle_Adj;
        double *d_obstacle_v;
        double *d_edges;
        int *d_member;

        hipMalloc((void **)&d_obstacle_A, obstacle_A_size);
        hipMalloc((void **)&d_obstacle_b, obstacle_b_size);
        hipMalloc((void **)&d_obstacle_Adj, obstacle_Adj_size);
        hipMalloc((void **)&d_obstacle_v, obstacle_v_size);
        hipMalloc((void **)&d_edges, edges_size);
        hipMalloc((void **)&d_member, member_size);

        // Prepare edge data
        double obstacle_A_flat[num_obstacles * 16];
        double obstacle_b_flat[num_obstacles * 4];
        double obstacle_Adj_flat[num_obstacles * 16];
        double obstacle_v_flat[num_obstacles * 8];
        double edges_flat[num_edges * 16];

        for (int o = 0; o < num_obstacles; o++) {
            // copy obstacle A
            for (int col = 0; col < 4; col++)
            {
                for (int row = 0; row < 4; row++)
                {
                    obstacle_A_flat[o * 16 + row + col*4] = obstacles[o].A(row, col);
                    obstacle_Adj_flat[o * 16 + row + col*4] = obstacles[o].Adjacency(row, col);
                }
                obstacle_b_flat[o * 4 + col] = obstacles[o].b(col);
            }
            for (int col = 0; col < 2; col++)
            {
                for (int row = 0; row < 4; row++)
                {
                    obstacle_v_flat[o * 8 + row + col*4] = obstacles[o].v(row, col);
                }
            }
        }
        // copy edges
        for (int i = 0; i < num_edges; i++)
        {
            Eigen::MatrixXd mat = edges[i];
            for (int col = 0; col < 4; col++)
            {
                for (int row = 0; row < 4; row++)
                {
                    edges_flat[i * 16 + row + col*4] = mat(row, col);
                }
            }
        }

        // hipEvent_t start, stop;
        // hipEventCreate(&start);
        // hipEventCreate(&stop);

        // Copy data to device
        hipMemcpy(d_obstacle_A, obstacle_A_flat, obstacle_A_size, hipMemcpyHostToDevice);
        hipMemcpy(d_obstacle_b, obstacle_b_flat, obstacle_b_size, hipMemcpyHostToDevice);
        hipMemcpy(d_obstacle_Adj, obstacle_Adj_flat, obstacle_Adj_size, hipMemcpyHostToDevice);
        hipMemcpy(d_obstacle_v, obstacle_v_flat, obstacle_v_size, hipMemcpyHostToDevice);
        // hipMemcpy(d_obstacle_A, obstacles[0].A.data(), obstacle_A_size, hipMemcpyHostToDevice);
        // hipMemcpy(d_obstacle_b, obstacles[0].b.data(), obstacle_b_size, hipMemcpyHostToDevice);
        // hipMemcpy(d_obstacle_Adj, obstacles[0].Adjacency.data(), obstacle_Adj_size, hipMemcpyHostToDevice);
        // hipMemcpy(d_obstacle_v, obstacles[0].v.data(), obstacle_v_size, hipMemcpyHostToDevice);
        hipMemcpy(d_edges, edges_flat, edges_size, hipMemcpyHostToDevice);

        // Launch the kernel
        int blockSize = 256;
        int gridSize = (num_obstacles * num_edges + blockSize - 1) / blockSize;

        // hipEventRecord(start);
        obstacleMembershipHeuristic<<<gridSize, blockSize>>>(d_obstacle_A, d_obstacle_b, d_obstacle_Adj, d_obstacle_v, d_edges, d_member, num_edges, num_obstacles);
        // hipEventRecord(stop);

        // Copy the result back to the host
        hipMemcpy(member.data(), d_member, member_size, hipMemcpyDeviceToHost);

        // hipEventSynchronize(stop);
        // float milliseconds = 0;
        // hipEventElapsedTime(&milliseconds, start, stop);
        // printf("That took: %f ms\n", milliseconds);

        // Free device memory
        hipFree(d_obstacle_A);
        hipFree(d_obstacle_b);
        hipFree(d_obstacle_Adj);
        hipFree(d_obstacle_v);
        hipFree(d_edges);
        hipFree(d_member);
    }
}

// void GraphQP::ObstacleMembershipHeuristic(Obstacle obstacle, const std::vector<matrix_t> edges, int_vector_t &member)
// {
//     // 0 if out, 1 if in, 2 if uncertain
//     // #pragma omp parallel for
//     for (int i = 0; i < edges.size(); i++) {
//         vector_t A_hyp_(2), A_hyp(2);
//         scalar_t b_hyp_, b_hyp;
//         matrix_t coll = (obstacle.A * edges[i]).colwise() - obstacle.b;
//         if (((coll.array() <= 0).colwise().all()).any()) {
//             member[i] = 1;
//         } else {
//             A_hyp.setZero();
//             b_hyp = 0;
//             A_hyp_.setZero();
//             b_hyp_ = 0;
//             for (int j = 0; j < edges[i].cols(); j++) {
//                 getSeparatingHyperplane(obstacle, edges[i].block(0,j,2, 1), A_hyp_, b_hyp_);
//                 A_hyp += A_hyp_;
//                 b_hyp += b_hyp_;
//             }
//             A_hyp /= edges[i].cols();
//             b_hyp /= edges[i].cols();

//             bool safe = (((A_hyp.transpose() * edges[i].block(0, 0, 2, edges[i].cols())).array() - b_hyp).array() >= 0).all();
//             if (safe == 1) {
//                 member[i] = 0;
//             } else {
//                 member[i] = 2;
//             }
//         }
//     }
// }

// void getSeparatingHyperplane(Obstacle obstacle, vector_t x, vector_t &A_hyp, scalar_t &b_hyp)
// {
//     int closest_point = -1;
//     scalar_t closest_dist = 1e3;
//     scalar_t dist_to_point;
//     Eigen::Array<bool, Eigen::Dynamic, 1> inds;

//     // (obstacle.v.block(0,0,obstacle.v.rows(),2).rowwise() - x.transpose()).rowwise().squaredNorm().minCoeff(&closest_point);

//     for (int j = 0; j < obstacle.v.rows(); j++) {
//         dist_to_point = (x - obstacle.v.block(j,0,1,2).transpose()).squaredNorm();
//         if (dist_to_point < closest_dist) {
//             closest_point = j;
//             closest_dist = dist_to_point;
//         }
//     }
//     vector_t faces = obstacle.Adjacency.block(closest_point,0,1,obstacle.Adjacency.cols()).transpose();
//     inds = (obstacle.A.block(0,0,obstacle.A.rows(),2) * x - obstacle.b).array() > -1e-2 && faces.array() > 0;
//     for (int j = 0; j < inds.size(); j++) {
//         if (inds(j) > 0) {
//             A_hyp += obstacle.A.block(j,0,1,2).transpose();
//         }
//     }
//     A_hyp = A_hyp / A_hyp.norm();
//     b_hyp = (A_hyp.transpose() * obstacle.v.block(closest_point,0,1,2).transpose()).value();
// }
