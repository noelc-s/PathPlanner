#include "hip/hip_runtime.h"
#include <kernel.hpp>
#include <Eigen/Core>

#include <iostream>
#include <stdio.h>

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

// CUDA Version
namespace Kernel
{
    __device__ void getSeparatingHyperplane(double *obstacle_A, double *obstacle_b, double *obstacle_Adj, double *obstacle_v, const double *x, double *A_hyp, double &b_hyp)
    {
        int closest_point = -1;
        double closest_dist = 1e3;
        double dist_to_point = 0.0;
        double temp[2] = {0.0};

        for (int j = 0; j < 4; j++) {
            dist_to_point = 0.0;
            for (int k = 0; k < 2; k++) {
                temp[k] = x[k] - obstacle_v[j + k*4];
                dist_to_point += temp[k] * temp[k];
            }
            if (dist_to_point < closest_dist) {
                closest_point = j;
                closest_dist = dist_to_point;
            }
        }

        const int num_faces = 4;
        for (int j = 0; j < num_faces; j++) {
            if ((obstacle_Adj[closest_point + j * 4] > 0) && (obstacle_A[j] * x[0] + obstacle_A[j + 4] * x[1] - obstacle_b[j] > -1e-2)) {
                for (int k = 0; k < 2; k++) {
                    A_hyp[k] += obstacle_A[j + 4*k];
                }
            }
        }

        double norm = sqrt(A_hyp[0] * A_hyp[0] + A_hyp[1] * A_hyp[1]);
        A_hyp[0] /= norm;
        A_hyp[1] /= norm;
        b_hyp = A_hyp[0] * obstacle_v[closest_point] + A_hyp[1] * obstacle_v[closest_point + 4];
    }

    __global__ void obstacleMembershipHeuristic(double *obstacle_A, double *obstacle_b, double *Adj, double *v, const double *edges, int *member, const int num_edges)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= num_edges)
            return;

        const double *edge = &edges[i * 16];

        bool in_obstacle = false;
        for (int k = 0; k < 4; k++) {
            bool result[4] = {false};
            for (int j = 0; j < 4; j++)
            {
                result[j] = (obstacle_A[j] * edge[4*k] + obstacle_A[j + 4] * edge[4*k+1] + obstacle_A[j + 8] * edge[4*k+2] + obstacle_A[j + 12] * edge[4*k+3] - obstacle_b[j]) <= 0;
            }
            if (result[0] & result[1] & result[2] & result[3])
            {
                in_obstacle = true;
                break;
            }
        }
        if (in_obstacle)
        {
            member[i] = 1;
        }
        else
        {
            // If not in obstacle, then we have to see if we have a separating hyperplane
            double A_hyp[2] = {0.0};
            double b_hyp = 0.0;

            for (int j = 0; j < 4; j++) {
                double A_hyp_[2] {0.0};
                double b_hyp_ = 0.0;
                getSeparatingHyperplane(obstacle_A, obstacle_b, Adj, v, &edge[j * 4], A_hyp_, b_hyp_);
                A_hyp[0] += A_hyp_[0];
                A_hyp[1] += A_hyp_[1];
                b_hyp += b_hyp_;
            }

            A_hyp[0] /= 4;
            A_hyp[1] /= 4;
            b_hyp /= 4;

            // printf("%f\n", A_hyp[0]);
            // printf("%f\n", A_hyp[1]);
            // printf("%f\n", b_hyp);

            bool safe = true;
            for (int j = 0; j < 4; j++) {
                double result = A_hyp[0] * edge[j * 4] + A_hyp[1] * edge[j * 4 + 1] - b_hyp;
                if (result < 0) {
                    safe = false;
                    break;
                }
            }

            member[i] = safe ? 0 : 2;

        }
    }

    void GraphQP_ObstacleMembershipHeuristic(Obstacle obstacle, const std::vector<Eigen::MatrixXd> &edges, std::vector<int> &member)
    {
        int num_edges = edges.size();
        int member_size = num_edges * sizeof(int);

        // Memory sizes
        size_t obstacle_A_size = 16 * sizeof(double);        // 4x4 matrix
        size_t obstacle_b_size = 4 * sizeof(double);         // 4x1 vector
        size_t obstacle_Adj_size = 16 * sizeof(double);         // 4x1 vector
        size_t obstacle_v_size = 8 * sizeof(double);         // 4x1 vector
        size_t edges_size = num_edges * 16 * sizeof(double); // num_edges x 4x4 matrix

        // Allocate memory on the device
        double *d_obstacle_A;
        double *d_obstacle_b;
        double *d_obstacle_Adj;
        double *d_obstacle_v;
        double *d_edges;
        int *d_member;

        hipMalloc((void **)&d_obstacle_A, obstacle_A_size);
        hipMalloc((void **)&d_obstacle_b, obstacle_b_size);
        hipMalloc((void **)&d_obstacle_Adj, obstacle_Adj_size);
        hipMalloc((void **)&d_obstacle_v, obstacle_v_size);
        hipMalloc((void **)&d_edges, edges_size);
        hipMalloc((void **)&d_member, member_size);

        // Prepare edge data
        double edges_flat[num_edges * 16];
        for (int i = 0; i < num_edges; i++)
        {
            Eigen::MatrixXd mat = edges[i];
            for (int col = 0; col < 4; col++)
            {
                for (int row = 0; row < 4; row++)
                {
                    edges_flat[i * 16 + row + col*4] = mat(row, col);
                }
            }
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Copy data to device
        hipMemcpy(d_obstacle_A, obstacle.A.data(), obstacle_A_size, hipMemcpyHostToDevice);
        hipMemcpy(d_obstacle_b, obstacle.b.data(), obstacle_b_size, hipMemcpyHostToDevice);
        hipMemcpy(d_obstacle_Adj, obstacle.Adjacency.data(), obstacle_Adj_size, hipMemcpyHostToDevice);
        hipMemcpy(d_obstacle_v, obstacle.v.data(), obstacle_v_size, hipMemcpyHostToDevice);
        hipMemcpy(d_obstacle_b, obstacle.b.data(), obstacle_b_size, hipMemcpyHostToDevice);
        hipMemcpy(d_edges, edges_flat, edges_size, hipMemcpyHostToDevice);

        // Launch the kernel
        int blockSize = 256;
        int gridSize = (num_edges + blockSize - 1) / blockSize;

        hipEventRecord(start);
        obstacleMembershipHeuristic<<<gridSize, blockSize>>>(d_obstacle_A, d_obstacle_b, d_obstacle_Adj, d_obstacle_v, d_edges, d_member, num_edges);
        hipEventRecord(stop);

        // Copy the result back to the host
        hipMemcpy(member.data(), d_member, member_size, hipMemcpyDeviceToHost);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("That took: %f ms\n", milliseconds);

        // Free device memory
        hipFree(d_obstacle_A);
        hipFree(d_obstacle_b);
        hipFree(d_obstacle_Adj);
        hipFree(d_obstacle_v);
        hipFree(d_edges);
        hipFree(d_member);
    }
}

// void GraphQP::ObstacleMembershipHeuristic(Obstacle obstacle, const std::vector<matrix_t> edges, int_vector_t &member)
// {
//     // 0 if out, 1 if in, 2 if uncertain
//     // #pragma omp parallel for
//     for (int i = 0; i < edges.size(); i++) {
//         vector_t A_hyp_(2), A_hyp(2);
//         double b_hyp_, b_hyp;
//         matrix_t coll = (obstacle.A * edges[i]).colwise() - obstacle.b;
//         if (((coll.array() <= 0).colwise().all()).any()) {
//             member[i] = 1;
//         } else {
//             A_hyp.setZero();
//             b_hyp = 0;
//             A_hyp_.setZero();
//             b_hyp_ = 0;
//             for (int j = 0; j < edges[i].cols(); j++) {
//                 getSeparatingHyperplane(obstacle, edges[i].block(0,j,2, 1), A_hyp_, b_hyp_);
//                 A_hyp += A_hyp_;
//                 b_hyp += b_hyp_;
//             }
//             A_hyp /= edges[i].cols();
//             b_hyp /= edges[i].cols();

//             bool safe = (((A_hyp.transpose() * edges[i].block(0, 0, 2, edges[i].cols())).array() - b_hyp).array() >= 0).all();
//             if (safe == 1) {
//                 member[i] = 0;
//             } else {
//                 member[i] = 2;
//             }
//         }
//     }
// }

// void getSeparatingHyperplane(Obstacle obstacle, vector_t x, vector_t &A_hyp, double &b_hyp)
// {
//     int closest_point = -1;
//     double closest_dist = 1e3;
//     double dist_to_point;
//     Eigen::Array<bool, Eigen::Dynamic, 1> inds;

//     // (obstacle.v.block(0,0,obstacle.v.rows(),2).rowwise() - x.transpose()).rowwise().squaredNorm().minCoeff(&closest_point);

//     for (int j = 0; j < obstacle.v.rows(); j++) {
//         dist_to_point = (x - obstacle.v.block(j,0,1,2).transpose()).squaredNorm();
//         if (dist_to_point < closest_dist) {
//             closest_point = j;
//             closest_dist = dist_to_point;
//         }
//     }
//     vector_t faces = obstacle.Adjacency.block(closest_point,0,1,obstacle.Adjacency.cols()).transpose();
//     inds = (obstacle.A.block(0,0,obstacle.A.rows(),2) * x - obstacle.b).array() > -1e-2 && faces.array() > 0;
//     for (int j = 0; j < inds.size(); j++) {
//         if (inds(j) > 0) {
//             A_hyp += obstacle.A.block(j,0,1,2).transpose();
//         }
//     }
//     A_hyp = A_hyp / A_hyp.norm();
//     b_hyp = (A_hyp.transpose() * obstacle.v.block(closest_point,0,1,2).transpose()).value();
// }
