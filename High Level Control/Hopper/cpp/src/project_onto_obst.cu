
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

__global__ void addMatrices(float *P_x, float *P_y, float *C, float *A_obs, float *b_obs, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        float A_i__x;
        float b_i;
        float in = 1;

        for (int i = 0; i < 4; i++)
        {
            A_i__x = A_obs[i * 2] * P_x[idx] + A_obs[i * 2 + 1] * P_y[idx];
            b_i = b_obs[i];
            if (A_i__x > b_i)
            {
                in = 0;
            }
        }

        C[idx] = in;
    }
}

hipEvent_t start, stop;

int main()
{

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-2, 2);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int N = 300;
    const int size = N * sizeof(float);

    // Host matrices
    float h_x[N], h_y[N], h_C[N];
    float A_O[8] = {-0.2577, 0.9662,
                    0.9545, -0.2983,
                    -0.9545, 0.2983,
                    0.2577, -0.9662};
    float b_O[4] = {0.7343, 0.5608, 0.7517, 0.6828};

    // Initialize matrices A and B
    for (int i = 0; i < N; ++i)
    {
        h_x[i] = static_cast<float>(dis(gen));
        h_y[i] = static_cast<float>(dis(gen));
    }

    // Device matrices
    float *d_A, *d_B, *d_C, *A_obs, *b_obs;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    hipMalloc((void **)&A_obs, 8 * sizeof(float));
    hipMalloc((void **)&b_obs, 4 * sizeof(float));

    // Copy matrices A and B to the device
    hipMemcpy(d_A, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_y, size, hipMemcpyHostToDevice);
    hipMemcpy(A_obs, A_O, 8 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_obs, b_O, 4 * sizeof(float), hipMemcpyHostToDevice);

    // Define the number of threads per block and the number of blocks
    int threadsPerBlock = 16;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    hipEventRecord(start);
    addMatrices<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, A_obs, b_obs, N);
    hipEventRecord(stop);

    // Copy the result matrix C back to the host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print the result
    for (int i = 0; i < N; ++i) {
        std::cout << h_x[i] << ", " << h_y[i] << ", " << h_C[i] << std::endl;
    }
    std::cout << std::endl;
    std::cout << "This took: " << milliseconds << "milliseconds" << std::endl;
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
